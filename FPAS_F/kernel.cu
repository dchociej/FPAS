#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_complex.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h":
#include <hipfft/hipfft.h>

#include <stdio.h>
#include "common\cpu_bitmap.h"
#include "common\book.h"

struct DataBlock {
	unsigned char   *dev_bitmap;
};



__global__ void copy2bitmap(hipComplex *in, unsigned char *ptr) {
	// Odwzorowanie z blockIdx na po�o�enie piksela
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;

	float aaa = (atan2(in[offset].y, in[offset].x));
	
	ptr[offset * 4 + 0] = 0; //2550000 * abs(in[offset].x);//(atan2(in[offset].y, in[offset].x)); //
	ptr[offset * 4 + 1] = 0;//2550000 * abs(in[offset].y);//;
	ptr[offset * 4 + 2] = 255* abs(aaa);
	ptr[offset * 4 + 3] = 255;
}



__global__ void shift2Dout(hipComplex *input, hipfftComplex *output)
{
    int i = threadIdx.x;
	int j = threadIdx.y;
	int n = blockIdx.x;
	int m = blockIdx.y;
	int di = blockDim.x / 2;
	int dj = blockDim.y / 2;

//	float *temp;
//	hipMalloc(temp, sizeof(float)*blockDim.x / 2 * blockDim.y / 2);
	if ((i < di) && (j < dj))
	output[(i+di) + (j+dj)*blockDim.x] = input[i + j*blockDim.x];
	if ((i >= di) && (j < dj))
	output[(i - di) + (j + dj)*blockDim.x] = input[i + j*blockDim.x];
	if ((i >= di) && (j >= dj))
	output[(i - di) + (j - dj)*blockDim.x] = input[i + j*blockDim.x];
	if ((i < di) && (j >= dj))
	output[(i + di) + (j - dj)*blockDim.x] = input[i + j*blockDim.x];

}


__device__ __forceinline__ hipComplex expf(hipComplex z)
{

	hipComplex res;
	float t = expf(z.x);
	sincosf(z.y, &res.y, &res.x);
	res.x *= t;
	res.y *= t;

	return res;

}

__global__ void calculate(hipComplex *fths, int *xo, int *yo, double *uo, float *zo2, float dfxs, float lambda, float k0, int Ts, float *fxs, float * y0seg, float* x0seg, int S_Bx, int S_By, int N_Bx, int N_By, int q)
{



	float yp = yo[threadIdx.x] - y0seg[blockIdx.y];

	float xp = xo[threadIdx.x] - x0seg[blockIdx.x];

    float rp = sqrt(zo2[threadIdx.x] + xp*xp + yp*yp);


    float inv_rp = 1 / rp;

    float fxp = xp*inv_rp / lambda;
	float fyp = yp*inv_rp / lambda;



	int iifx = round(fxp / dfxs) + S_Bx / 2 + 1;
	int iify = round(fyp / dfxs) + S_By / 2 + 1;

	if (iifx <= 0 || iifx > S_Bx || iify <= 0 || iify > S_Bx){
		iifx = S_Bx / 2 + 1;
		iify = S_Bx / 2 + 1;
	}



	hipComplex c0;
	hipComplex arg;
	hipComplex arg1;
//	arg.x = (k0*rp - 2 * HIP_PI_F*(fxs[iifx] + fxs[iify])*(Ts / 2));

//	arg1.x = (2 * HIP_PI_F  * uo[threadIdx.x] / 6400); 

//	arg1.x = 2;
//	arg.x = -35.699;
	hipComplex res;
	float t = arg1.x*inv_rp;
	sincosf(-arg.x, &res.y, &res.x);
	res.x *= t;
	res.y *= t;

//	c0 = expf(arg);
//	hipComplex uoo = expf(arg1);
//	c0.x = uo[threadIdx.x] * c0.x;
//	c0.y = uo[threadIdx.x] * c0.y;
	c0 = res;


	


	//fths[iifx + blockIdx.x*S_Bx + iify*S_Bx*N_Bx + blockIdx.y* S_Bx*N_Bx*S_By].x += c0.x;
	//fths[iifx + blockIdx.x*S_Bx + iify*S_Bx*N_Bx + blockIdx.y* S_Bx*N_Bx*S_By].y += c0.y;

	fths[iifx + iify*S_Bx + blockIdx.y*S_Bx*S_By + blockIdx.x* S_Bx*N_Bx*S_By].x += c0.x;
	fths[iifx + iify*S_Bx + blockIdx.y*S_Bx*S_By + blockIdx.x* S_Bx*N_Bx*S_By].y += c0.y;

//	fths[iifx + iify*S_Bx + blockIdx.x*S_Bx*S_By + blockIdx.y* S_Bx*N_Bx*S_By].x = 128;
//	fths[iifx + iify*S_Bx + blockIdx.x*S_Bx*S_By + blockIdx.y* S_Bx*N_Bx*S_By].y = 128 ;

}


hipfftResult preparePlan2D(hipfftHandle* plan, int nRows, int nCols, int batch){

	int n[2] = { nRows, nCols };

	hipfftResult result = hipfftPlanMany(plan,
		2, //rank
		n, //dimensions = {nRows, nCols}
		0, //inembed
		batch, //istride
		1, //idist
		0, //onembed
		batch, //ostride
		1, //odist
		HIPFFT_C2C, //hipfftType
		batch /*batch*/);

	if (result != 0){

		//		std::cout << "preparePlan2D error, result: " << result << "/n";
		return result;
	}
	return result;
}

hipfftResult execute2D(hipfftHandle* plan, hipfftComplex* idata, hipfftComplex* odata, int direction){

	hipfftResult result = hipfftExecC2C(*plan, idata, odata, direction);

	if (result != 0){

		//		cout << "execute2D error, result: " << result << "/n";
		return result;
	}
	return result;
}




void FPAS_CGH_2D(int Np, int* xo, int* yo, int* zo, double* uo, int Nx, int Ny, int dx, float lambda, int S_Bx, int S_By, int q, hipComplex* fths_p, hipComplex* fths_s)
{
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	double k0 = 2 * HIP_PI_F / lambda;

	int x_size = (Nx / 2) + ((Nx / 2) - 1) + 1;
	int y_size = (Ny / 2) + ((Ny / 2) - 1) + 1;

	float *x = (float*)malloc(x_size * sizeof(float));
	float *y = (float*)malloc(y_size * sizeof(float));

	for (int t = 0; t < x_size; t++){
		x[t] = (-Nx / 2 + t)*dx;
	}

	for (int t = 0; t < y_size; t++){
		y[t] = (-Ny / 2 + t)*dx;
	}

	int N_Bx = Nx / S_Bx; // doda� obs�ug� nie ca�kowitych dziele�
	int N_By = Ny / S_By;

	int Ts = S_Bx*dx;

	float dfxs = 1 / (float)Ts;

	int fxs_size = (S_Bx / 2) + ((S_Bx / 2) - 1) + 1;
	float *fxs = (float*)malloc(fxs_size * sizeof(float));

	for (int t = 0; t < fxs_size; t++){
		fxs[t] = (float)(-S_Bx / 2 + t)*dfxs;
	}

	float * x0seg = (float*)malloc((N_Bx)* sizeof(float));

	for (int t = 0; t < N_By; t++){
		x0seg[t] = (x[0] + (t*Ts) + Ts / 2);
	}

	float * y0seg = (float*)malloc((N_By)* sizeof(float));

	for (int t = 0; t < N_By; t++){
		y0seg[t] = (y[0] + (t*Ts) + Ts / 2);
	}


	float *z02;
	z02 = (float*)malloc((Np)* sizeof(float));

	for (int t = 0; t < Np; t++)
		z02[t] = zo[t] * zo[t];


	int *d_xo;
	int *d_yo;
	float *d_z02;
	double *d_uo;

	hipMalloc((void**)&d_xo, sizeof(int)*Np);
	hipMalloc((void**)&d_yo, sizeof(int)*Np);
	hipMalloc((void**)&d_z02, sizeof(float)*Np);
	hipMalloc((void**)&d_uo, sizeof(double)*Np);

	hipMemcpy(d_xo, xo, Np*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_yo, yo, Np*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_z02, z02, Np*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_uo, uo, Np*sizeof(double), hipMemcpyHostToDevice);

    float *d_fxs;
	float *d_y0seg;
	float *d_x0seg;

	hipMalloc((void**)&d_x0seg, sizeof(float)*N_Bx);
	hipMalloc((void**)&d_y0seg, sizeof(float)*N_By);
	hipMalloc((void**)&d_fxs, sizeof(float)*fxs_size);

	hipMemcpy(d_fxs, fxs, fxs_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x0seg, x0seg, N_Bx*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y0seg, y0seg, N_By*sizeof(float), hipMemcpyHostToDevice);

	dim3 grid;
	grid.x = N_Bx;//y
	grid.y = N_By;//x

	dim3 block;
	block.x = Np; //z
	block.y = 1;

	hipEventRecord(start, 0);
	calculate << < grid, block >> >(fths_p, d_xo, d_yo, d_uo, d_z02, dfxs, lambda, k0, Ts, d_fxs, d_y0seg, d_x0seg, S_Bx, S_Bx, N_Bx, N_By, q);


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Time for the kernel: %f ms\n", time);


	dim3 grids;
	grids.x = N_Bx;
	grids.y = N_By;

	dim3 blocks;
	blocks.x = S_Bx;
	blocks.y = S_By;

	hipEventRecord(start, 0);
	shift2Dout << < grids, blocks >> > (fths_p, fths_s);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	printf("Time for the kernel: %f ms\n", time);

	/*	hipComplex *host;
	host = (hipComplex*)malloc(sizeof(hipComplex)*Nosx*Nosy*Np);
	hipMemcpy(host, fths, sizeof(hipComplex)*Nosx*Nosy*Np, hipMemcpyDeviceToHost);
	*/
}

int main()
{

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/*START CUDA CALC PART - DEKLARACJE*/
	int Nx = 1024;
	int Ny = 1024;
	int dx = 8;
	float lambda = 0.5;
	/*START CUDA FFT 2D PART - DEKLARACJE*/
	int S_Bx = 16;
	int S_By = 16;

	hipfftComplex* h_out; //dane wynikowe CPU
	hipfftComplex* holo; //dane wyj�ciowe GPU

	int batch = Nx / S_Bx * Ny / S_By;  //N_Bx*N_By
	hipfftHandle forwardPlan;

	preparePlan2D(&forwardPlan, S_Bx, S_By, batch);


	h_out = (hipfftComplex*)malloc(sizeof(hipfftComplex)*S_Bx*S_By*batch); //allokacja pami�ci na wynik (CPU)

	hipMalloc(&holo, sizeof(hipfftComplex) *S_Bx*S_By*batch); //allokacja pami�ci na dane wyj�ciowe (GPU)
	hipMemset(holo, 0, sizeof(hipfftComplex)*S_Bx*S_By*batch); //Wype�nianie zaalokowanej pami�ci zerami (GPU)

	/*END CUDA FFT 2D PART - DEKLARACJE*/

	/*Kod kernela*/
	int Np = 1024;

	int *xo;
	int *yo;
	int *zo;
	double *uo;


	xo = (int*)malloc((Np)* sizeof(int));
	yo = (int*)malloc((Np)* sizeof(int));
	zo = (int*)malloc((Np)* sizeof(int));
	uo = (double*)malloc((Np)* sizeof(double));

	for (int tt = 0; tt < Np; tt++)
	{
		xo[tt] = tt;
		yo[tt] = tt;
		zo[tt] = tt;
	}



	double W = 0.1e3;
	double dxo = W / 10.0;
	int foo = 0;

	for (foo = 0; foo < Np; foo++)
	{
	xo[foo] = W;
	yo[foo] = W;
	zo[foo] = 500e3;
	uo[foo] = 3.14;
	}
//	uo = exp(2 * pi * 1i * rand(1, Np) / 6400); % object point phase - random




	hipComplex *fths_p;  
	hipComplex *fths_s;

	hipfftComplex* fhs;

	//	hipMalloc(&fhs, sizeof(hipfftComplex)*S_Bx*S_By*batch); //allokacja pami�ci na dane wej�ciowe (GPU)
	hipMalloc(&fths_p, sizeof(hipComplex)*Nx*Ny);
	hipMemset(fths_p, 0, sizeof(hipComplex)*Nx*Ny);

	hipMalloc(&fths_s, sizeof(hipComplex)*Nx*Ny);
	hipMemset(fths_s, 0, sizeof(hipComplex)*Nx*Ny);

	hipEventRecord(start, 0);
	/*START CUDA CALC PART */
	FPAS_CGH_2D(Np, xo, yo, zo, uo, Nx, Ny, dx, lambda, S_Bx, S_By, 2, fths_p, fths_s);

	hipMemcpy(h_out, fths_p, sizeof(hipfftComplex)*S_Bx*S_By*batch, hipMemcpyDeviceToHost);

	/*START CUDA FFT PART */
	execute2D(&forwardPlan, fths_p, holo, HIPFFT_FORWARD);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	/*Wyswietlanie modulu/fazy*/
//	hipMemcpy(h_out, holo, sizeof(hipfftComplex)*S_Bx*S_By*batch, hipMemcpyDeviceToHost);

	
	/*END CUDA FFT PART */

	// Retrieve result from device and store it in host array
	hipEventElapsedTime(&time, start, stop);
	printf("Time for the kernel: %f ms\n", time);
	printf("Time for the kernel: %f ms\n", h_out[213100].x);

	printf("END \n");


		DataBlock   data;
		CPUBitmap bitmap(Nx, Ny, &data);
		unsigned char    *dev_bitmap;

		HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, bitmap.image_size()));
		data.dev_bitmap = dev_bitmap;
		hipMemset(dev_bitmap, 255, bitmap.image_size());

		dim3    grid(Nx, Ny);
		copy2bitmap << <grid, 1 >> >(fths_p, dev_bitmap);
	
		HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap,
			bitmap.image_size(),
			hipMemcpyDeviceToHost));

		bitmap.display_and_exit();
	
	return 0;
}
